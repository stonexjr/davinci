#include "oit_cuda_kernel.cuh"
#include <thrust/scan.h>
#include <thrust/fill.h>
#include <thrust/device_vector.h>
#include <hip/hip_math_constants.h>
#include <hip/hip_vector_types.h>
#include <hip/hip_vector_types.h>

extern "C"
{
    //int* data: device address of data
    void computePrefixSum(int* dSrcData, int* dDstData, uint count)
    {
        thrust::device_ptr<int> dSrcPtr(dSrcData);
        thrust::device_ptr<int> dDstPtr(dDstData);
        thrust::exclusive_scan(dSrcPtr, dSrcPtr + count, dDstPtr);
    }

    void cudaFill(int *data, int count, int val)
    {
        thrust::device_ptr<int> dPtr(data);
        thrust::fill(dPtr, dPtr + count, (int)val);
    }

}

